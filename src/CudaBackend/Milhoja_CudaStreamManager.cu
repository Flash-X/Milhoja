#include "hip/hip_runtime.h"
#include "Milhoja_CudaStreamManager.h"

#include <cassert>
#include <iostream>
#include <stdexcept>

#include "Milhoja.h"
#include "Milhoja_Logger.h"

#ifdef MILHOJA_OPENACC_OFFLOADING
#include <openacc.h>
#endif

namespace milhoja {

bool   CudaStreamManager::initialized_ = false;
bool   CudaStreamManager::finalized_   = false;
int    CudaStreamManager::nMaxStreams_ = -1;

/**
 * Instantiate and initialize the application's singleton stream manager object.
 *
 * \param nMaxStreams - the maximum number of streams to be made available.  The
 *                      given value must be a positive integer.
 */
void   CudaStreamManager::initialize(const int nMaxStreams) {
    // finalized_ => initialized_
    // Therefore, no need to check finalized_.
    if (initialized_) {
        throw std::logic_error("[CudaStreamManager::initialize] "
                               "Already initialized");
    } else if (nMaxStreams <= 0) {
        // We need at least one stream to avoid deadlocking in requestStream
        // when there are no free streams.
        throw std::invalid_argument("[CudaStreamManager::initialize] "
                                    "Need at least one stream");
    }

    // Create/initialize
    nMaxStreams_ = nMaxStreams;
    initialized_ = true;

    instance();
}

/**
 * 
 */
void    CudaStreamManager::finalize(void) {
    if        (!initialized_) {
        throw std::logic_error("[CudaStreamManager::finalize] Never initialized");
    } else if (finalized_) {
        throw std::logic_error("[CudaStreamManager::finalize] Already finalized");
    }

    Logger::instance().log("[CudaStreamManager] Finalizing ...");

    pthread_mutex_lock(&idxMutex_);

    if (streams_.size() != nMaxStreams_) {
        std::string   errMsg =   "[CudaStreamManager::finalize] "
                               + std::to_string(nMaxStreams_ - streams_.size()) 
                               + " out of "
                               + std::to_string(nMaxStreams_)
                               + " streams have not been released";
        throw std::runtime_error(errMsg);
    }

#ifdef MILHOJA_OPENACC_OFFLOADING
    Logger::instance().log(  "[CudaStreamManager] No longer using "
                           + std::to_string(streams_.size())
                           + " CUDA streams/OpenACC asynchronous queues");
#else
    hipError_t   cErr = hipErrorInvalidValue;
    for (std::size_t i=0; i<streams_.size(); ++i) {
         cErr = hipStreamDestroy(streams_[i].cudaStream);
         if (cErr != hipSuccess) {
            std::string  errMsg = "[CudaStreamManager::finalize] ";
            errMsg += "Unable to destroy CUDA stream\n";
            errMsg += "CUDA error - " + std::string(hipGetErrorName(cErr)) + "\n";
            errMsg += std::string(hipGetErrorString(cErr));
            throw std::runtime_error(errMsg);
         }
    }
    Logger::instance().log(  "[CudaStreamManager] Destroyed "
                           + std::to_string(streams_.size())
                           + " CUDA streams");
#endif

    pthread_mutex_unlock(&idxMutex_);

    pthread_cond_destroy(&streamReleased_);
    pthread_mutex_destroy(&idxMutex_);

    nMaxStreams_ = -1;

    finalized_ = true;

    Logger::instance().log("[CudaStreamManager] Finalized");
}

/**
 * Request access to the singleton stream manager.  Before calling this routine,
 * calling code must first initialize the manager.
 */
CudaStreamManager&   CudaStreamManager::instance(void) {
    if        (!initialized_) {
        throw std::logic_error("[CudaStreamManager::instance] Singleton not initialized");
    } else if (finalized_) {
        throw std::logic_error("[CudaStreamManager::instance] No access after finalization");
    }

    static CudaStreamManager   manager;
    return manager;
}

/**
 * 
 *
 * \return 
 */
CudaStreamManager::CudaStreamManager(void)
    : streams_{}
{
    Logger::instance().log("[CudaStreamManager] Initializing...");
    assert(streams_.size() == 0);

    pthread_cond_init(&streamReleased_, NULL);
    pthread_mutex_init(&idxMutex_, NULL);

    pthread_mutex_lock(&idxMutex_);

#ifdef MILHOJA_OPENACC_OFFLOADING
    Stream         stream{};
    for (int i=0; i<nMaxStreams_; ++i) {
         stream.accAsyncQueue = i + 1;
         stream.cudaStream = static_cast<hipStream_t>(acc_get_cuda_stream(stream.accAsyncQueue));
         if (stream.cudaStream == nullptr) {
            std::string  errMsg = "[CudaStreamManager::CudaStreamManager] ";
            errMsg += "CUDA stream is null\n";
            throw std::runtime_error(errMsg);
         }

         streams_.push_back( std::move(stream) );
    }
    Logger::instance().log(  "[CudaStreamManager] Acquired " 
                           + std::to_string(streams_.size())
                           + " CUDA streams from OpenACC asynchronous queues");
#else
    Stream   stream{};
    for (std::size_t i=0; i<nMaxStreams_; ++i) {
         hipError_t    cErr = hipStreamCreate(&(stream.cudaStream));
         if (cErr != hipSuccess) {
            std::string  errMsg = "[CudaStreamManager::CudaStreamManager] ";
            errMsg += "Unable to create CUDA stream\n";
            errMsg += "CUDA error - " + std::string(hipGetErrorName(cErr)) + "\n";
            errMsg += std::string(hipGetErrorString(cErr)) + "\n";
            pthread_mutex_unlock(&idxMutex_);
            throw std::runtime_error(errMsg);
         }

         streams_.push_back( std::move(stream) );
    }
    Logger::instance().log(  "[CudaStreamManager] Created "
                           + std::to_string(streams_.size())
                           + " CUDA streams");
#endif

    Logger::instance().log("[CudaStreamManager] Created and ready for use");

    pthread_mutex_unlock(&idxMutex_);
}

/**
 * Based on the StreamManager's singleton design pattern, this will only be
 * called at program termination.
 */
CudaStreamManager::~CudaStreamManager(void) {
    if (initialized_ && !finalized_) {
        std::cerr << "[CudaStreamManager::~CudaStreamManager] ERROR - Not finalized"
                  << std::endl;
    }
}

/**
 * 
 *
 * \return 
 */
int  CudaStreamManager::numberFreeStreams(void) {
    pthread_mutex_lock(&idxMutex_);
    std::size_t nStreams = streams_.size();
    pthread_mutex_unlock(&idxMutex_);

    if (nStreams > INT_MAX) {
        std::string  errMsg = "[CudaStreamManager::numberFreeStreams] ";
        errMsg += "Too many streams created\n";
        throw std::overflow_error(errMsg);
    }

    return static_cast<int>(nStreams);
}

/**
 * This should be by move only!
 *
 * Refer to the documentation of the requestStream member function of the
 * RuntimeBackend class.
 *
 * \todo   If, as is the case for CUDA, the streams are relatively cheap
 *         objects, then should we allow this routine to allocate more streams
 *         rather than block?  This could be helpful to prevent possible
 *         deadlocks when a code needs to acquire more than one stream.
 *         However, we shouldn't make this a feature of the general design
 *         concept unless we know that all stream managers can dynamically grow
 *         their reserve.  This, presently, cannot be implemented as
 *         OpenACC+CUDA with PGI on Summit has an upper limit of 32 streams.
 * \todo   Add in logging of release if verbosity level is high enough.
 */
Stream    CudaStreamManager::requestStream(const bool block) {
    // Get exclusive access to the free stream queue so that we can safely get
    // the ID of a free stream from it.  It is also important for the case when
    // we need to wait for a streamReleased signal.  In particular, we need to
    // know that no thread can emit that signal between when we acquire the
    // mutex and when we begin to wait for the signal.  In other words, we won't
    // miss the signal, which could be important if there is presently only one
    // stream in use by client code.
    pthread_mutex_lock(&idxMutex_);

    if (streams_.size() <= 0) {
        if (block) {
            // Block until a stream is released and this thread hits the 
            // lottery and actually gets to take control of it.
            //
            // To avoid deadlocking on this wait, we need at least one stream
            // out for business.  Given that we know that the number of free
            // streams is zero, there can be no deadlock so long as this object
            // is managing at least one stream.
            //
            // There exists another possibility for deadlocking based on the
            // notion that any code can request a stream.  Consider the case of
            // 5 streams total and five data packets that request and receive
            // one stream each.  If each action associated with the data packets
            // subsequently request an extra stream, then they will all wait for
            // a free stream and none will therefore release a stream.
            do {
                Logger::instance().log("[CudaStreamManager] No streams available.  Blocking as requested.");
                pthread_cond_wait(&streamReleased_, &idxMutex_);
//                Logger::instance().log("[CudaStreamManager] Stream has been released");
            } while(streams_.size() <= 0);
        } else {
            Logger::instance().log("[CudaStreamManager] No streams available. Returning null stream as requested.");
            pthread_mutex_unlock(&idxMutex_);

            return Stream{};
        }
    }

    Stream   stream = std::move(streams_.front());
    streams_.pop_front();

    pthread_mutex_unlock(&idxMutex_);

    return stream;
}

/**
 * Refer to the documentation of the releaseStream member function of the
 * RuntimeBackend class.
 */
void   CudaStreamManager::releaseStream(Stream& stream) {
    if (stream.cudaStream == nullptr) {
        throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                    "Given stream has null CUDA stream");
#ifdef MILHOJA_OPENACC_OFFLOADING
    } else if (stream.accAsyncQueue == NULL_ACC_ASYNC_QUEUE) {
        throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                    "Given stream has null OpenACC asynchronous queue");
#endif
    }

    pthread_mutex_lock(&idxMutex_);

    if (streams_.size() >= nMaxStreams_) {
        pthread_mutex_unlock(&idxMutex_);
        throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                    "All streams accounted for.  No streams to release.");
    }

#ifdef DEBUG_RUNTIME
    // Streams will be released frequently and we might have a great many
    // streams.  Therefore, we don't want to perform this error checking by
    // default.
    for (const auto& freeStream : streams_) {
        if (stream.cudaStream == freeStream.cudaStream) {
            pthread_mutex_unlock(&idxMutex_);
            throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                        "Given stream is already free");
        }
    }
#endif

    // We must put the stream back in the queue before emitting the signal
    streams_.push_back( std::move(stream) );
    pthread_cond_signal(&streamReleased_);

    pthread_mutex_unlock(&idxMutex_);
}

}

