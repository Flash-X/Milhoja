#include "hip/hip_runtime.h"
#include "Milhoja_CudaBackend.h"

#include "Milhoja_Logger.h"
#include "Milhoja_CudaGpuEnvironment.h"
#include "Milhoja_CudaStreamManager.h"
#include "Milhoja_CudaMemoryManager.h"

namespace milhoja {

/**
 * Initialize a CudaBackend object and all the helpers that it relies on.  It
 * is intended that this only ever be called by the RuntimeBackend's initialize
 * member function.  In this way, the initialized object should be a singleton.
 */
CudaBackend::CudaBackend(const unsigned int nStreams,
                         const std::size_t  nBytesInMemoryPools) {
    Logger::instance().log("[CudaBackend] Initializing...");

    // Since RuntimeBackend calls instance() inside initialize() and this constructor
    // should only be called once, these lines effectively carry out the 
    // initialize() work of this derived class.
    CudaGpuEnvironment::initialize();
    CudaStreamManager::initialize(nStreams);
    CudaMemoryManager::initialize(nBytesInMemoryPools);

    Logger::instance().log("[CudaBackend] Created and ready for use");
}

/**
 * Refer to the RuntimeBackend documentation for more information.
 */
void    CudaBackend::finalize(void) {
    Logger::instance().log("[CudaBackend] Finalizing...");

    CudaMemoryManager::instance().finalize();
    CudaStreamManager::instance().finalize();
    CudaGpuEnvironment::instance().finalize();

    RuntimeBackend::finalize();

    Logger::instance().log("[CudaBackend] Finalized");
}


/**
 * Refer to the RuntimeBackend documentation for more information.
 */
int  CudaBackend::maxNumberStreams(void) const {
    return CudaStreamManager::instance().maxNumberStreams();
}

/**
 * Refer to the RuntimeBackend documentation for more information.
 */
int  CudaBackend::numberFreeStreams(void) {
    return CudaStreamManager::instance().numberFreeStreams();
}

/**
 * Refer to the RuntimeBackend documentation for more information.
 */
Stream    CudaBackend::requestStream(const bool block) {
    return CudaStreamManager::instance().requestStream(block);
}

/**
 * Refer to the RuntimeBackend documentation for more information.
 */
void      CudaBackend::releaseStream(Stream& stream) {
    CudaStreamManager::instance().releaseStream(stream);
}

/**
 * Refer to the RuntimeBackend documentation for more information.
 */
void  CudaBackend::initiateHostToGpuTransfer(DataPacket& packet) {
    // One and only one thread should own this packet at any given time.
    // Therefore, it has exclusive access and this code is thread-safe.
    hipError_t cErr = hipMemcpyAsync(packet.copyToGpuStart_gpu(),
                                       packet.copyToGpuStart_host(),
                                       packet.copyToGpuSizeInBytes(),
                                       hipMemcpyHostToDevice,
                                       packet.stream());
    if (cErr != hipSuccess) {
        std::string  errMsg = "[CudaBackend::initiateHostToGpuTransfer] ";
        errMsg += "Unable to initiate H-to-Gpu transfer\n";
        errMsg += "CUDA error - " + std::string(hipGetErrorName(cErr)) + "\n";
        errMsg += std::string(hipGetErrorString(cErr)) + "\n";
        throw std::runtime_error(errMsg);
    }
}

/**
 *  Refer to the RuntimeBackend documentation for general information.
 *
 *  Initiate an asychronous transfer of the packet from the device to the host
 *  on the packet's stream.  As part of this, launch on the same stream the given
 *  callback for handling the unpacking and other auxiliary work that must occur
 *  once the packet is back in pinned memory.
 *
 * \param  packet   - the data packet to transfer.
 * \param  callback - the routine that will be registered with the CUDA runtime
 *                    so that the routine can unpack the packet (likely using
 *                    unpack) and perform other desired actions.
 * \param  callbackData - the data that must be passed to the callback so that
 *                        it can carry out its work.  This resource just passes
 *                        through this routine so that this routine has no
 *                        responsibility in managing the resources.
 */
void  CudaBackend::initiateGpuToHostTransfer(DataPacket& packet,
                                             GPU_TO_HOST_CALLBACK_FCN callback,
                                             void* callbackData) {
    // One and only one thread should own this packet at any given time.
    // Therefore, it has exclusive access and this code is thread-safe.
    hipStream_t  stream = packet.stream();

    hipError_t   cErr = hipMemcpyAsync(packet.returnToHostStart_host(),
                                         packet.returnToHostStart_gpu(),
                                         packet.returnToHostSizeInBytes(),
                                         hipMemcpyDeviceToHost,
                                         stream);
    if (cErr != hipSuccess) {
        std::string  errMsg = "[CudaBackend::initiateGpuToHostTransfer] ";
        errMsg += "Unable to initiate Gpu-to-H transfer\n";
        errMsg += "CUDA error - " + std::string(hipGetErrorName(cErr)) + "\n";
        errMsg += std::string(hipGetErrorString(cErr)) + "\n";
        throw std::runtime_error(errMsg);
    }

    cErr = hipLaunchHostFunc(stream, callback, callbackData); 
    if (cErr != hipSuccess) {
        std::string  errMsg = "[CudaBackend::initiateGpuToHostTransfer] ";
        errMsg += "Unable to register Gpu-to-H callback function\n";
        errMsg += "CUDA error - " + std::string(hipGetErrorName(cErr)) + "\n";
        errMsg += std::string(hipGetErrorString(cErr)) + "\n";
        throw std::runtime_error(errMsg);
    }
}

/**
 * Refer to the RuntimeBackend documentation for more information.
 */
void      CudaBackend::requestGpuMemory(const std::size_t pinnedBytes,
                                        void** pinnedPtr,
                                        const std::size_t gpuBytes,
                                        void** gpuPtr) {
    CudaMemoryManager::instance().requestMemory(pinnedBytes, pinnedPtr,
                                                gpuBytes, gpuPtr);
}

/**
 * Refer to the RuntimeBackend documentation for more information.
 */
void      CudaBackend::releaseGpuMemory(void** pinnedPtr, void** gpuPtr) {
    CudaMemoryManager::instance().releaseMemory(pinnedPtr, gpuPtr);
}

/**
 * Refer to the RuntimeBackend documentation for more information.
 *
 */
void      CudaBackend::reset(void) {
    CudaMemoryManager::instance().reset();
}

}

