#include "hip/hip_runtime.h"
#include "CudaRuntime.h"

#include <stdexcept>
#include <iostream>

#include "ThreadTeam.h"
#include "Grid.h"
#include "OrchestrationLogger.h"

#ifdef USE_CUDA_BACKEND
#include "CudaGpuEnvironment.h"
#include "CudaStreamManager.h"
#include "CudaMemoryManager.h"
#include "CudaDataPacket.h"
#endif

#include "Flash.h"

namespace orchestration {

unsigned int    CudaRuntime::nTeams_            = 0;
unsigned int    CudaRuntime::maxThreadsPerTeam_ = 0;
bool            CudaRuntime::instantiated_      = false;

/**
 * 
 *
 * \return 
 */
CudaRuntime& CudaRuntime::instance(void) {
    static CudaRuntime     orSingleton;
    return orSingleton;
}

/**
 * 
 *
 * \return 
 */
void CudaRuntime::setLogFilename(const std::string& filename) {
    orchestration::Logger::setLogFilename(filename);
}

/**
 * 
 *
 * \return 
 */
void CudaRuntime::setNumberThreadTeams(const unsigned int nTeams) {
    if (instantiated_) {
        throw std::logic_error("[CudaRuntime::setNumberThreadTeams] "
                               "Set only when runtime does not exist");
    } else if (nTeams == 0) {
        throw std::invalid_argument("[CudaRuntime::setNumberThreadTeams] "
                                    "Need at least one ThreadTeam");
    }

    nTeams_ = nTeams;
}

/**
 * 
 *
 * \return 
 */
void CudaRuntime::setMaxThreadsPerTeam(const unsigned int nThreads) {
    if (instantiated_) {
        throw std::logic_error("[CudaRuntime::setMaxThreadsPerTeam] "
                               "Set only when runtime does not exist");
    } else if (nThreads == 0) {
        throw std::invalid_argument("[CudaRuntime::setMaxThreadsPerTeam] "
                                    "Need at least one thread per team");
    }

    maxThreadsPerTeam_ = nThreads;
}

/**
 * 
 *
 * \return 
 */
CudaRuntime::CudaRuntime(void)
    : teams_{nullptr}
{
    Logger::instance().log("[CudaRuntime] Initializing...");

    if (nTeams_ <= 0) {
        throw std::invalid_argument("[CudaRuntime::CudaRuntime] "
                                    "Need to create at least one team");
    }

    teams_ = new ThreadTeam*[nTeams_];
    for (unsigned int i=0; i<nTeams_; ++i) {
        teams_[i] = new ThreadTeam(maxThreadsPerTeam_, i);
    }

#ifdef USE_CUDA_BACKEND
    CudaGpuEnvironment&    gpuEnv = CudaGpuEnvironment::instance();
    std::string   msg =   "[Runtime] " 
                        + std::to_string(gpuEnv.nGpuDevices()) 
                        + " GPU device(s) per process found\n"
                        + gpuEnv.information();
    Logger::instance().log(msg);

    CudaStreamManager::instance();
    CudaMemoryManager::instance();
#endif

    instantiated_ = true;

    Logger::instance().log("[CudaRuntime] Created and ready for use");
}

/**
 * 
 *
 * \return 
 */
CudaRuntime::~CudaRuntime(void) {
    Logger::instance().log("[CudaRuntime] Finalizing...");

    instantiated_ = false;

    for (unsigned int i=0; i<nTeams_; ++i) {
        delete teams_[i];
        teams_[i] = nullptr;
    }
    delete [] teams_;
    teams_ = nullptr;

    Logger::instance().log("[CudaRuntime] Finalized");
}

/**
 * 
 *
 * \return 
 */
void CudaRuntime::executeCpuTasks(const std::string& actionName,
                                  const RuntimeAction& cpuAction) {
    Logger::instance().log("[Runtime] Start single CPU action");

    if (cpuAction.teamType != ThreadTeamDataType::BLOCK) {
        throw std::logic_error("[CudaRuntime::executeCpuTasks] "
                               "Given CPU action should run on block-based "
                               "thread team, which is not in configuration");
    } else if (cpuAction.nTilesPerPacket != 0) {
        throw std::invalid_argument("[CudaRuntime::executeCpuTasks] "
                                    "CPU tiles/packet should be zero since it is tile-based");
    } else if (nTeams_ < 1) {
        throw std::logic_error("[CudaRuntime::executeCpuTasks] "
                               "Need at least one ThreadTeam in runtime");
    }

    //***** ASSEMBLE THREAD TEAM CONFIGURATION
    // CPU action parallel pipeline
    // 1) CPU action applied to blocks by CPU team
    ThreadTeam*   cpuTeam = teams_[0];

    //***** START EXECUTION CYCLE
    cpuTeam->startCycle(cpuAction, "CPU_Block_Team");

    //***** ACTION PARALLEL DISTRIBUTOR
    unsigned int   level = 0;
    Grid&   grid = Grid::instance();
    for (auto ti = grid.buildTileIter(level); ti->isValid(); ti->next()) {
        cpuTeam->enqueue( ti->buildCurrentTile() );
    }
    cpuTeam->closeQueue();

    // host thread blocks until cycle ends
    cpuTeam->wait();

    // No need to break apart the thread team configuration

    Logger::instance().log("[CudaRuntime] End single CPU action");
}

/**
 * 
 *
 * \return 
 */
#if defined(USE_CUDA_BACKEND)
void CudaRuntime::executeGpuTasks(const std::string& bundleName,
                                  const RuntimeAction& gpuAction) {
    Logger::instance().log("[Runtime] Start single GPU action");

    if (gpuAction.teamType != ThreadTeamDataType::SET_OF_BLOCKS) {
        throw std::logic_error("[CudaRuntime::executeGpuTasks] "
                               "Given GPU action should run on a thread team "
                               "that works with data packets of blocks");
    } else if (gpuAction.nTilesPerPacket <= 0) {
        throw std::invalid_argument("[CudaRuntime::executeGpuTasks] "
                                    "Need at least one tile per packet");
    } else if (nTeams_ < 1) {
        throw std::logic_error("[CudaRuntime::executeGpuTasks] "
                               "Need at least one ThreadTeam in runtime");
    }

    //***** ASSEMBLE THREAD TEAM CONFIGURATION
    // GPU action parallel pipeline
    // 1) Asynchronous transfer of Packets of Blocks to GPU
    // 2) GPU action applied to blocks in packet by GPU team
    // 3) Mover/Unpacker transfers packet back to CPU and
    //    copies results to Grid data structures
    ThreadTeam*       gpuTeam   = teams_[0];
    gpuTeam->attachDataReceiver(&gpuToHost_);

    //***** START EXECUTION CYCLE
    gpuTeam->startCycle(gpuAction, "GPU_PacketOfBlocks_Team");

    //***** ACTION PARALLEL DISTRIBUTOR

    unsigned int   level = 0;
    Grid&          grid = Grid::instance();
    auto           packet_gpu = std::shared_ptr<DataPacket>{};
    assert(packet_gpu == nullptr);
    assert(packet_gpu.use_count() == 0);
    for (auto ti = grid.buildTileIter(level); ti->isValid(); ti->next()) {
        packet_gpu = std::shared_ptr<DataPacket>{ new CudaDataPacket{ ti->buildCurrentTile() } };
        packet_gpu->initiateHostToDeviceTransfer();

        gpuTeam->enqueue( std::move(packet_gpu) );
        assert(packet_gpu == nullptr);
        assert(packet_gpu.use_count() == 0);
    }

    gpuTeam->closeQueue();

    // host thread blocks until cycle ends
    gpuTeam->wait();

    //***** BREAK APART THREAD TEAM CONFIGURATION
    gpuTeam->detachDataReceiver();

    Logger::instance().log("[CudaRuntime] End single GPU action");
}
#endif

/**
 * 
 *
 * \return 
 */
#if defined(USE_CUDA_BACKEND)
void CudaRuntime::executeTasks_FullPacket(const std::string& bundleName,
                                          const RuntimeAction& cpuAction,
                                          const RuntimeAction& gpuAction,
                                          const RuntimeAction& postGpuAction) {
    Logger::instance().log("[Runtime] Start CPU/GPU/Post-GPU action bundle");

    if        (cpuAction.teamType != ThreadTeamDataType::BLOCK) {
        throw std::logic_error("[CudaRuntime::executeTasks_FullPacket] "
                               "Given CPU action should run on tile-based "
                               "thread team, which is not in configuration");
    } else if (cpuAction.nTilesPerPacket != 0) {
        throw std::invalid_argument("[CudaRuntime::executeTasks_FullPacket] "
                                    "CPU tiles/packet should be zero since it is tile-based");
    } else if (gpuAction.teamType != ThreadTeamDataType::SET_OF_BLOCKS) {
        throw std::logic_error("[CudaRuntime::executeTasks_FullPacket] "
                               "Given GPU action should run on packet-based "
                               "thread team, which is not in configuration");
    } else if (gpuAction.nTilesPerPacket <= 0) {
        throw std::invalid_argument("[CudaRuntime::executeTasks_FullPacket] "
                                    "Need at least one tile per GPU packet");
    } else if (postGpuAction.teamType != ThreadTeamDataType::BLOCK) {
        throw std::logic_error("[CudaRuntime::executeTasks_FullPacket] "
                               "Given post-GPU action should run on tile-based "
                               "thread team, which is not in configuration");
    } else if (postGpuAction.nTilesPerPacket != 0) {
        throw std::invalid_argument("[CudaRuntime::executeTasks_FullPacket] "
                                    "Post-GPU should have zero tiles/packet as "
                                    "client code cannot control this");
    } else if (nTeams_ < 3) {
        throw std::logic_error("[CudaRuntime::executeTasks_FullPacket] "
                               "Need at least three ThreadTeams in runtime");
    }

    //***** ASSEMBLE THREAD TEAM CONFIGURATION
    // CPU action parallel pipeline
    // 1) CPU action applied to blocks by CPU team
    //
    // GPU/Post-GPU action parallel pipeline
    // 1) Asynchronous transfer of Packets of Blocks to GPU
    // 2) GPU action applied to blocks in packet by GPU team
    // 3) Mover/Unpacker transfers packet back to CPU,
    //    copies results to Grid data structures, and
    //    pushes blocks to Post-GPU team
    // 4) Post-GPU action applied by host via Post-GPU team
    ThreadTeam*        cpuTeam     = teams_[0];
    ThreadTeam*        gpuTeam     = teams_[1];
    ThreadTeam*        postGpuTeam = teams_[2];

    cpuTeam->attachThreadReceiver(postGpuTeam);
    gpuTeam->attachThreadReceiver(postGpuTeam);
    gpuTeam->attachDataReceiver(&gpuToHost_);
    gpuToHost_.attachDataReceiver(postGpuTeam);

    unsigned int nTotalThreads =       cpuAction.nInitialThreads
                                 +     gpuAction.nInitialThreads
                                 + postGpuAction.nInitialThreads;
    if (nTotalThreads > postGpuTeam->nMaximumThreads()) {
        throw std::logic_error("[CudaRuntime::executeTasks_FullPacket] "
                                "Post-GPU could receive too many thread "
                                "activation calls from CPU and GPU teams");
    }

    //***** START EXECUTION CYCLE
    cpuTeam->startCycle(cpuAction, "Concurrent_CPU_Block_Team");
    gpuTeam->startCycle(gpuAction, "Concurrent_GPU_Packet_Team");
    postGpuTeam->startCycle(postGpuAction, "Post_GPU_Block_Team");

    //***** ACTION PARALLEL DISTRIBUTOR
    unsigned int                      level = 0;
    Grid&                             grid = Grid::instance();
    std::shared_ptr<Tile>             tile_cpu{};
    std::shared_ptr<Tile>             tile_gpu{};
    std::shared_ptr<DataPacket>       packet_gpu = std::shared_ptr<DataPacket>{};
    for (auto ti = grid.buildTileIter(level); ti->isValid(); ti->next()) {
        // If we create a first shared_ptr and enqueue it with one team, it is
        // possible that this shared_ptr could have the action applied to its
        // data and go out of scope before we create a second shared_ptr.  In
        // this case, the data item's resources would be released prematurely.
        // To avoid this, we create all copies up front and before enqueing any
        // copy.
        tile_cpu = ti->buildCurrentTile();
        tile_gpu = tile_cpu;
        if (   (tile_cpu.get() != tile_gpu.get())
            || (tile_cpu.use_count() != 2)) {
            throw std::runtime_error("tile_cpu and tile_gpu not matched");
        }

        packet_gpu = std::shared_ptr<DataPacket>{ new CudaDataPacket{std::move(tile_gpu)} };
        if (   (tile_gpu != nullptr)
            || (tile_gpu.use_count() != 0)) {
            throw std::runtime_error("tile_gpu not nulled");
//        } else if (   (packet_gpu->getTile().get() != tile_cpu.get())
//                   || (tile_cpu.use_count() != 2)) {
//            throw std::runtime_error("tile_cpu and packet_gpu not matched");
        }

        // CPU action parallel pipeline
        cpuTeam->enqueue( std::move(tile_cpu) );
        if (   (tile_cpu != nullptr)
            || (tile_cpu.use_count() != 0)) {
            throw std::runtime_error("tile_cpu not nulled");
        }

        // GPU/Post-GPU action parallel pipeline
        packet_gpu->initiateHostToDeviceTransfer();
        gpuTeam->enqueue( std::move(packet_gpu) );
        if (   (packet_gpu != nullptr)
            || (packet_gpu.use_count() != 0)) {
            throw std::runtime_error("packet_gpu not nulled");
        }
    }
    gpuTeam->closeQueue();
    cpuTeam->closeQueue();

    // host thread blocks until cycle ends
    cpuTeam->wait();
    gpuTeam->wait();
    postGpuTeam->wait();

    //***** BREAK APART THREAD TEAM CONFIGURATION
    cpuTeam->detachThreadReceiver();
    gpuTeam->detachThreadReceiver();
    gpuTeam->detachDataReceiver();
    gpuToHost_.detachDataReceiver();

    Logger::instance().log("[CudaRuntime] End CPU/GPU action");
}
#endif

}

