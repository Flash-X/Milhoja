#include "CudaMoverUnpacker.h"

#include <cassert>

namespace orchestration {

void CudaMoverUnpacker::increaseThreadCount(const unsigned int nThreads) {
    throw std::logic_error("[CudaMoverUnpacker::increaseThreadCount] "
                           "CudaMoverUnpackers do no have threads to awaken");
}

void CudaMoverUnpacker::enqueue(std::shared_ptr<DataItem>&& packet) {
    // Bring data back to host.  Use asynchronous transfer so that we can keep
    // the transfer off the default stream and therefore only wait on this
    // transfer.
    hipStream_t  stream = *(packet->stream().object);
    hipError_t   cErr = hipMemcpyAsync(packet->hostPointer(), packet->gpuPointer(),
                                         packet->sizeInBytes(),
                                         hipMemcpyDeviceToHost, stream);
    if (cErr != hipSuccess) {
        std::string  errMsg = "[CudaMoverUnpacker::enqueue] ";
        errMsg += "Unable to execute D-to-H transfer\n";
        errMsg += "CUDA error - " + std::string(hipGetErrorName(cErr)) + "\n";
        errMsg += std::string(hipGetErrorString(cErr)) + "\n";
        throw std::runtime_error(errMsg);
    }
    hipStreamSynchronize(stream);

    // Undo the packet => data transferred back to location in Grid's data
    // structures
    packet->unpack(); 

    // Transfer the ownership of the data item in the packet to the next team
    if (dataReceiver_) {
        dataReceiver_->enqueue(packet->getTile());
    }

    // This function must take over control of the packet from the calling code.
    // In this case, the data packet is now no longer needed.
    // TODO: Is this necessary and correct?
    packet.reset();
    assert(packet == nullptr);
    assert(packet.use_count() == 0);
}

void CudaMoverUnpacker::closeQueue(void) {
    if (dataReceiver_) {
        dataReceiver_->closeQueue();
    }
}

}

