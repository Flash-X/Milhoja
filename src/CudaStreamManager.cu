#include "hip/hip_runtime.h"
#ifndef USE_CUDA_BACKEND
#error "This file need not be compiled if the CUDA backend isn't used"
#endif

#include "CudaStreamManager.h"

#include <cassert>
#include <iostream>
#include <stdexcept>

#ifdef ENABLE_OPENACC_OFFLOAD
#include <openacc.h>
#endif

#include "OrchestrationLogger.h"

namespace orchestration {

bool   CudaStreamManager::instantiated_ = false;
int    CudaStreamManager::nMaxStreams_ = -1;

/**
 * Instantiate and initalize the application's singleton stream manager object.
 *
 * \param nMaxStreams - the maximum number of streams to be made available.  The
 *                      given value must be a positive integer.
 */
void   CudaStreamManager::instantiate(const int nMaxStreams) {
    if (instantiated_) {
        throw std::logic_error("[CudaStreamManager::instantiate] "
                               "Already instantiated");
    } else if (nMaxStreams <= 0) {
        // We need at least one stream to avoid deadlocking in requestStream
        // when there are no free streams.
        throw std::invalid_argument("[CudaStreamManager::instantiate] "
                                    "Need at least one stream");
    }

    // Create/initialize
    nMaxStreams_ = nMaxStreams;
    instantiated_ = true;

    instance();
}

/**
 * Before calling this routine, client code must first instantiate the manager.
 *
 * \return A reference to the stream manager for the associated
 *         runtime backend.
 */
CudaStreamManager&   CudaStreamManager::instance(void) {
    if (!instantiated_) {
        throw std::logic_error("[CudaStreamManager::instance] "
                               "CudaStreamManager must be instantiated first");
    }

    static CudaStreamManager   manager;
    return manager;
}

/**
 * 
 *
 * \return 
 */
CudaStreamManager::CudaStreamManager(void)
    : streams_{}
{
    Logger::instance().log("[CudaStreamManager] Initializing...");
    assert(streams_.size() == 0);

    pthread_cond_init(&streamReleased_, NULL);
    pthread_mutex_init(&idxMutex_, NULL);

    pthread_mutex_lock(&idxMutex_);

#ifdef ENABLE_OPENACC_OFFLOAD
    Stream         stream{};
    for (int i=0; i<nMaxStreams_; ++i) {
         stream.accAsyncQueue = i + 1;
         stream.cudaStream = static_cast<hipStream_t>(acc_get_cuda_stream(stream.accAsyncQueue));

         streams_.push_back( std::move(stream) );
    }
    Logger::instance().log(  "[CudaStreamManager] Acquired " 
                           + std::to_string(streams_.size())
                           + " CUDA streams from OpenACC asynchronous queues");
#else
    Stream   stream{};
    for (std::size_t i=0; i<nMaxStreams_; ++i) {
         hipError_t    cErr = hipStreamCreate(&(stream.cudaStream));
         if (cErr != hipSuccess) {
            std::string  errMsg = "[CudaStreamManager::CudaStreamManager] ";
            errMsg += "Unable to create CUDA stream\n";
            errMsg += "CUDA error - " + std::string(hipGetErrorName(cErr)) + "\n";
            errMsg += std::string(hipGetErrorString(cErr)) + "\n";
            pthread_mutex_unlock(&idxMutex_);
            throw std::runtime_error(errMsg);
         }

         streams_.push_back( std::move(stream) );
    }
    Logger::instance().log(  "[CudaStreamManager] Created "
                           + std::to_string(streams_.size())
                           + " CUDA streams");
#endif

    Logger::instance().log("[CudaStreamManager] Created and ready for use");

    pthread_mutex_unlock(&idxMutex_);
}

/**
 * Based on the StreamManager's singleton design pattern, this will only be
 * called at program termination.
 */
CudaStreamManager::~CudaStreamManager(void) {
    Logger::instance().log("[CudaStreamManager] Finalizing...");

    pthread_mutex_lock(&idxMutex_);

    // TODO: When designing an appropriate error handling system, should we
    // include the possibility of including warnings?  Should there be a logging
    // system to which we could write this?  Should the logging system patch
    // into a logging system offered by client code?
    if (streams_.size() != nMaxStreams_) {
        std::cerr << "[CudaStreamManager::~CudaStreamManager] WARNING - "
                  << (nMaxStreams_ - streams_.size()) 
                  << " out of " << nMaxStreams_
                  << " streams have not been released" << std::endl;
    }

#ifdef ENABLE_OPENACC_OFFLOAD
    Logger::instance().log(  "[CudaStreamManager] No longer using "
                           + std::to_string(streams_.size())
                           + " CUDA streams/OpenACC asynchronous queues");
#else
    hipError_t   cErr = hipErrorInvalidValue;
    for (std::size_t i=0; i<streams_.size(); ++i) {
         cErr = hipStreamDestroy(streams_[i].cudaStream);
         if (cErr != hipSuccess) {
            std::string  errMsg = "[CudaStreamManager::~CudaStreamManager] ";
            errMsg += "Unable to destroy CUDA stream\n";
            errMsg += "CUDA error - " + std::string(hipGetErrorName(cErr)) + "\n";
            errMsg += std::string(hipGetErrorString(cErr)) + "\n";
            std::cerr << errMsg;
         }
    }
    Logger::instance().log(  "[CudaStreamManager] Destroyed "
                           + std::to_string(streams_.size())
                           + " CUDA streams");
#endif

    pthread_mutex_unlock(&idxMutex_);

    pthread_cond_destroy(&streamReleased_);
    pthread_mutex_destroy(&idxMutex_);

    nMaxStreams_ = -1;
    instantiated_ = false;

    Logger::instance().log("[CudaStreamManager] Destroyed");
}

/**
 * 
 *
 * \return 
 */
int  CudaStreamManager::numberFreeStreams(void) {
    pthread_mutex_lock(&idxMutex_);
    std::size_t nStreams = streams_.size();
    pthread_mutex_unlock(&idxMutex_);

    if (nStreams > INT_MAX) {
        std::string  errMsg = "[CudaStreamManager::numberFreeStreams] ";
        errMsg += "Too many streams created\n";
        throw std::overflow_error(errMsg);
    }

    return static_cast<int>(nStreams);
}

/**
 * This should be by move only!
 *
 * Refer to the documentation of the requestStream member function of the
 * Backend class.
 *
 * \todo   If, as is the case for CUDA, the streams are relatively cheap
 *         objects, then should we allow this routine to allocate more streams
 *         rather than block?  This could be helpful to prevent possible
 *         deadlocks when a code needs to acquire more than one stream.
 *         However, we shouldn't make this a feature of the general design
 *         concept unless we know that all stream managers can dynamically grow
 *         their reserve.  This, presently, cannot be implemented as
 *         OpenACC+CUDA with PGI on Summit has an upper limit of 32 streams.
 * \todo   Add in logging of release if verbosity level is high enough.
 */
Stream    CudaStreamManager::requestStream(const bool block) {
    // Get exclusive access to the free stream queue so that we can safely get
    // the ID of a free stream from it.  It is also important for the case when
    // we need to wait for a streamReleased signal.  In particular, we need to
    // know that no thread can emit that signal between when we acquire the
    // mutex and when we begin to wait for the signal.  In other words, we won't
    // miss the signal, which could be important if there is presently only one
    // stream in use by client code.
    pthread_mutex_lock(&idxMutex_);

    if (streams_.size() <= 0) {
        if (block) {
            // Block until a stream is released and this thread hits the 
            // lottery and actually gets to take control of it.
            //
            // To avoid deadlocking on this wait, we need at least one stream
            // out for business.  Given that we know that the number of free
            // streams is zero, there can be no deadlock so long as this object
            // is managing at least one stream.
            //
            // There exists another possibility for deadlocking based on the
            // notion that any code can request a stream.  Consider the case of
            // 5 streams total and five data packets that request and receive
            // one stream each.  If each action associated with the data packets
            // subsequently request an extra stream, then they will all wait for
            // a free stream and none will therefore release a stream.
            do {
                Logger::instance().log("[CudaStreamManager] No streams available.  Blocking as requested.");
                pthread_cond_wait(&streamReleased_, &idxMutex_);
//                Logger::instance().log("[CudaStreamManager] Stream has been released");
            } while(streams_.size() <= 0);
        } else {
            Logger::instance().log("[CudaStreamManager] No streams available. Returning null stream as requested.");
            pthread_mutex_unlock(&idxMutex_);

            return Stream{};
        }
    }

    Stream   stream = std::move(streams_.front());
    streams_.pop_front();

    pthread_mutex_unlock(&idxMutex_);

    return stream;
}

/**
 * Refer to the documentation of the releaseStream member function of the
 * Backend class.
 */
void   CudaStreamManager::releaseStream(Stream& stream) {
    if (stream.cudaStream == nullptr) {
        throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                    "Given stream has null CUDA stream");
#ifdef ENABLE_OPENACC_OFFLOAD
    } else if (stream.accAsyncQueue == NULL_ACC_ASYNC_QUEUE) {
        throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                    "Given stream has null OpenACC asynchronous queue");
#endif
    }

    pthread_mutex_lock(&idxMutex_);

    if (streams_.size() >= nMaxStreams_) {
        pthread_mutex_unlock(&idxMutex_);
        throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                    "All streams accounted for.  No streams to release.");
    }

#ifdef DEBUG_RUNTIME
    // Streams will be released frequently and we might have a great many
    // streams.  Therefore, we don't want to perform this error checking by
    // default.
    for (const auto& freeStream : freeStreams_) {
        if (stream.cudaStream == freeStream.cudaStream) {
            pthread_mutex_unlock(&idxMutex_);
            throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                        "Given stream is already free");
        }
    }
#endif

    // We must put the stream back in the queue before emitting the signal
    streams_.push_back( std::move(stream) );
    pthread_cond_signal(&streamReleased_);

    pthread_mutex_unlock(&idxMutex_);
}

}

