#include "hip/hip_runtime.h"
#include "CudaStreamManager.h"

#include <cassert>
#include <iostream>
#include <stdexcept>

#include <openacc.h>

#include "OrchestrationLogger.h"

namespace orchestration {

// Default value chosen in conjunction with the error checking in the
// constructor such that client code will get an error if they do not explicitly
// set the numer of streams before accessing the manager.
int    CudaStreamManager::nMaxStreams_     = -1;
bool   CudaStreamManager::wasInstantiated_ = false;

/**
 * Before calling this routine, client code must first set the number of streams
 * to be managed using setMaxNumberStreams().
 *
 * \return 
 */
CudaStreamManager&   CudaStreamManager::instance(void) {
    static CudaStreamManager   stream_manager;
    Logger::instance().log("[CudaStreamManager] Access given to manager");
    return stream_manager;
}

/**
 * This member must be called before accessing the manager, but cannot be called
 * after accessing the manager.
 *
 * \return 
 */
void CudaStreamManager::setMaxNumberStreams(const int nMaxStreams) {
    if (wasInstantiated_) {
        throw std::logic_error("[CudaStreamManager::setMaxNumberStreams] "
                               "Cannot be set once the manager has been accessed");
    } else if (nMaxStreams <= 0) {
        // We need at least one stream to avoid deadlocking in requestStream
        // when there are no free streams.
        throw std::invalid_argument("[CudaStreamManager::setMaxNumberStreams] "
                                    "Need at least one stream");
    }

    nMaxStreams_ = nMaxStreams;
    Logger::instance().log( "[CudaStreamManager] Number of streams set to "
                           + std::to_string(nMaxStreams_));
}

/**
 * 
 *
 * \return 
 */
CudaStreamManager::CudaStreamManager(void)
    : streams_{nMaxStreams_},
      freeStreams_{}
{
    if (nMaxStreams_ <= 0) {
        throw std::invalid_argument("[CudaStreamManager::CudaStreamManager] "
                                    "Set max number of streams before accessing manager");
    } else if (streams_.size() > INT_MAX) {
        std::string  errMsg = "[CudaStreamManager::CudaStreamManager] ";
        errMsg += "Too many streams created\n";
        throw std::overflow_error(errMsg);
    }
    assert(freeStreams_.size() == 0);

    pthread_cond_init(&streamReleased_, NULL);
    pthread_mutex_init(&idxMutex_, NULL);
    Logger::instance().log("[CudaStreamManager] streamReleased condition variable initalized");
    Logger::instance().log("[CudaStreamManager] Free stream mutex initialized");

    pthread_mutex_lock(&idxMutex_);

    hipError_t   cErr = hipErrorInvalidValue;
    for (int i=0; i<streams_.size(); ++i) {
         cErr = hipStreamCreate(&(streams_[i]));
         if (cErr != hipSuccess) {
            std::string  errMsg = "[CudaStreamManager::CudaStreamManager] ";
            errMsg += "Unable to create CUDA stream\n";
            errMsg += "CUDA error - " + std::string(hipGetErrorName(cErr)) + "\n";
            errMsg += std::string(hipGetErrorString(cErr)) + "\n";
            pthread_mutex_unlock(&idxMutex_);
            throw std::runtime_error(errMsg);
         }

         // Make stream indices 1-based so that 0 can work as NULL_STREAM
         int   streamId = i + 1;
         // For some unknown reason, I need to call get before calling set.
         // If I don't do this, then the queue-stream linking doesn't happen
         // on the first block.
         void*   queue = acc_get_cuda_stream(streamId);
         acc_set_cuda_stream(streamId, streams_[i]);
         queue = acc_get_cuda_stream(streamId);
         if (queue != streams_[i]) {
            std::string  errMsg = "[CudaStreamManager::CudaStreamManager] ";
            errMsg += "OpenACC async queue not linked to CUDA Stream\n";
            pthread_mutex_unlock(&idxMutex_);
            throw std::runtime_error(errMsg);
         }

         freeStreams_.push_back( CudaStream(streamId, &(streams_[i])) );
    }
    Logger::instance().log(  "[CudaStreamManager] Created " 
                           + std::to_string(streams_.size())
                           + " CUDA streams");
    Logger::instance().log("[CudaStreamManager] Created and ready for use");

    wasInstantiated_ = true;

    pthread_mutex_unlock(&idxMutex_);
}

/**
 * 
 *
 * \return 
 */
CudaStreamManager::~CudaStreamManager(void) {
    pthread_mutex_lock(&idxMutex_);

    // TODO: When designing an appropriate error handling system, should we
    // include the possibility of including warnings?  Should there be a logging
    // system to which we could write this?  Should the logging system patch
    // into a logging system offered by client code?
    if (freeStreams_.size() != streams_.size()) {
        std::cerr << "[CudaStreamManager::~CudaStreamManager] WARNING - "
                  << (streams_.size() - freeStreams_.size()) 
                  << " out of " << streams_.size()
                  << " streams have not been released" << std::endl;
    }

    hipError_t   cErr = hipErrorInvalidValue;
    for (std::size_t i=0; i<streams_.size(); ++i) {
         cErr = hipStreamDestroy(streams_[i]);
         if (cErr != hipSuccess) {
            std::string  errMsg = "[CudaStreamManager::~CudaStreamManager] ";
            errMsg += "Unable to destroy CUDA stream\n";
            errMsg += "CUDA error - " + std::string(hipGetErrorName(cErr)) + "\n";
            errMsg += std::string(hipGetErrorString(cErr)) + "\n";
            std::cerr << errMsg;
         }
    }
    Logger::instance().log(  "[CudaStreamManager] Destroyed "
                           + std::to_string(streams_.size())
                           + " CUDA streams");

    pthread_mutex_unlock(&idxMutex_);

    pthread_cond_destroy(&streamReleased_);
    pthread_mutex_destroy(&idxMutex_);

    Logger::instance().log("[CudaStreamManager] Stream released condition variable destroyed");
    Logger::instance().log("[CudaStreamManager] Free stream mutex destroyed");

    wasInstantiated_ = false;
    Logger::instance().log("[CudaStreamManager] Destroyed");
}

/**
 * 
 *
 * \return 
 */
int  CudaStreamManager::numberFreeStreams(void) {
    pthread_mutex_lock(&idxMutex_);
    std::size_t nStreams = freeStreams_.size();
    pthread_mutex_unlock(&idxMutex_);

    if (nStreams > INT_MAX) {
        std::string  errMsg = "[CudaStreamManager::numberFreeStreams] ";
        errMsg += "Too many streams created\n";
        throw std::overflow_error(errMsg);
    }

    return static_cast<int>(nStreams);
}

/**
 * This should be by move only!
 *
 * If block is true and there are no free streams, this member function will
 * block the calling thread until a stream becomes free.
 *
 * \return The free stream that has been given to the calling code for exclusive
 * use.  If block is set to false and there are no free streams, then a null
 * stream object is returned.
 */
CudaStream    CudaStreamManager::requestStream(const bool block) {
    Logger::instance().log("[CudaStreamManager] Stream requested");

    // Get exclusive access to the free stream queue so that we can safely get
    // the ID of a free stream from it.  It is also important for the case when
    // we need to wait for a streamReleased signal.  In particular, we need to
    // know that no thread can emit that signal between when we acquire the
    // mutex and when we begin to wait for the signal.  In other words, we won't
    // miss the signal, which could be important if there is presently only one
    // stream in use by client code.
    pthread_mutex_lock(&idxMutex_);

    if (freeStreams_.size() <= 0) {
        if (block) {
            // Block until a stream is released and this thread hits the 
            // lottery and actually gets to take control of it.
            //
            // To avoid deadlocking on this wait, we need at least one stream
            // out for business.  Given that we know that the number of free
            // streams is zero, there can be no deadlock so long as this object
            // is managing at least one stream.
            do {
                Logger::instance().log("[CudaStreamManager] No streams available.  Blocking as requested.");
                pthread_cond_wait(&streamReleased_, &idxMutex_);
                Logger::instance().log("[CudaStreamManager] Stream has been released");
            } while(freeStreams_.size() <= 0);
        } else {
            Logger::instance().log("[CudaStreamManager] No streams available. Returning null stream as requested.");
            pthread_mutex_unlock(&idxMutex_);

            return CudaStream{};
        }
    }

    // The stream IDs are 1-based, but the queue is 0-based
    CudaStream   stream = std::move(freeStreams_.front());
    freeStreams_.pop_front();

    pthread_mutex_unlock(&idxMutex_);

    if ( stream.object != &(streams_[stream.id-1]) ) {
        throw std::invalid_argument("[CudaStreamManager::requestStream] "
                                    "Given stream ID and pointer not properly matched");
    }

    Logger::instance().log(  "[CudaStreamManager] Stream " 
                           + std::to_string(stream.id)
                           + " distributed");
    return stream;
}

/**
 * 
 *
 * \return 
 */
void   CudaStreamManager::releaseStream(CudaStream& stream) {
    if        (stream.id == CudaStream::NULL_STREAM_ID) {
        throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                    "Given stream has null ID");
    } else if (stream.id > streams_.size()) {
        throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                    "Stream ID is too large");
    } else if (stream.object == nullptr) {
        throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                    "Given stream has null object");
    } else if (stream.object != &(streams_[stream.id-1]) ) {
        throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                    "Given stream ID and pointer not properly matched");
    }

    pthread_mutex_lock(&idxMutex_);

    if (freeStreams_.size() >= streams_.size()) {
        pthread_mutex_unlock(&idxMutex_);
        throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                    "All streams accounted for.  No streams to release.");
    }

    for (const auto& freeStream : freeStreams_) {
        if (stream.id == freeStream.id) {
            pthread_mutex_unlock(&idxMutex_);
            throw std::invalid_argument("[CudaStreamManager::releaseStream] "
                                        "Given stream is already free");
        }
    }

    Logger::instance().log(  "[CudaStreamManager] Stream " 
                           + std::to_string(stream.id) 
                            + " released");

    // We must put the stream back in the queue before emitting the signal
    freeStreams_.push_back( std::move(stream) );
    pthread_cond_signal(&streamReleased_);

    pthread_mutex_unlock(&idxMutex_);
}

}

