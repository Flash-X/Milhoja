#include "hip/hip_runtime.h"
#include "CudaGpuEnvironment.h"

#include <stdexcept>
#include <sstream>

namespace orchestration {

bool  CudaGpuEnvironment::instantiated_ = false;

/**
 * 
 *
 * \return 
 */
CudaGpuEnvironment& CudaGpuEnvironment::instance(void) {
    static CudaGpuEnvironment     orSingleton;
    return orSingleton;
}

/**
 * 
 *
 * \return 
 */
CudaGpuEnvironment::CudaGpuEnvironment(void)
    : nDevices_{0},
      gpuDeviceName_{""},
      gpuCompMajor_{-1},
      gpuCompMinor_{-1},
      gpuMaxGridSize_{-1, -1, -1},
      gpuMaxThreadDim_{-1, -1, -1},
      gpuMaxThreadsPerBlock_{-1},
      gpuWarpSize_{-1},
      gpuClockRateHz_{-1.0},
      gpuMemClockRateHz_{-1.0},
      gpuMemBusWidthBytes_{-1},
      gpuTotalGlobalMemBytes_{0},
      gpuL2CacheSizeBytes_{-1},
      gpuSupportsL1Caching_{false},
      gpuNumMultiprocessors_{-1},
      gpuMaxConcurrentKernels_{-1}
{
    hipGetDeviceCount(&nDevices_);
    if (nDevices_ != 1) {
        throw std::runtime_error("[CudaGpuEnvironment::CudaGpuEnvironment] "
                                 "We insist upon 1 GPU per MPI task");
    }

    hipDeviceProp_t  prop;
    hipGetDeviceProperties(&prop, 0);

    if (prop.concurrentKernels != 1) {
        throw std::runtime_error("[CudaGpuEnvironment::CudaGpuEnvironment] "
                                 "GPU kernel concurrency is required");
    }

    gpuDeviceName_          = std::string(prop.name);
    gpuCompMajor_           = prop.major;
    gpuCompMinor_           = prop.minor;
    gpuMaxThreadsPerBlock_  = prop.maxThreadsPerBlock;
    gpuWarpSize_            = prop.warpSize;
    gpuClockRateHz_         = prop.clockRate * 1000;
    gpuMemClockRateHz_      = prop.memoryClockRate * 1000;
    gpuMemBusWidthBytes_    = round(prop.memoryBusWidth * 0.125);
    gpuTotalGlobalMemBytes_ = prop.totalGlobalMem;
    gpuL2CacheSizeBytes_    = prop.l2CacheSize;
    gpuSupportsL1Caching_   = (prop.localL1CacheSupported == 1);
    gpuNumMultiprocessors_  = prop.multiProcessorCount;
    for (unsigned int i=0; i<3; ++i) {
        gpuMaxGridSize_[i]  = prop.maxGridSize[i];
        gpuMaxThreadDim_[i] = prop.maxThreadsDim[i];
    }

    if (gpuCompMajor_ != 7) {
        throw std::runtime_error("[CudaGpuEnvironment::CudaGpuEnvironment] "
                                 "We assume GPU compute capability 7.X");
    }

    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications__technical-specifications-per-compute-capability
    gpuMaxConcurrentKernels_ = 128;

    instantiated_ = true;
}

/**
 * 
 *
 * \return 
 */
CudaGpuEnvironment::~CudaGpuEnvironment(void) {
    instantiated_ = false;
}

std::string  CudaGpuEnvironment::information(void) const {
    std::stringstream     info;
    info << "  Name                    "
         <<  gpuDeviceName_ << "\n";
    info << "  Clock Rate              "
         << (gpuClockRateHz_ * 1.0e-9) << " GHz\n";
    info << "  Memory Clock Rate       "
         << (gpuMemClockRateHz_ * 1.0e-9) << " GHz\n";
    info << "  Memory Bus Width        "
         <<  gpuMemBusWidthBytes_ << " bytes\n";
    info << "  Total Global Memory     "
         << (gpuTotalGlobalMemBytes_ / std::pow(1024.0, 3.0)) << " GB\n";
    info << "  L2 Cache Size           "
         << (gpuL2CacheSizeBytes_ / std::pow(1024.0, 2.0)) << " MB\n";
    info << "  Supports local L1 Cache "
         <<  (gpuSupportsL1Caching_ ? 'T' : 'F') << "\n";
    info << "  Compute Capability      "
         <<  gpuCompMajor_ << "." << gpuCompMinor_ << "\n";
    info << "  Max Grid Size           "
         <<  gpuMaxGridSize_[0] << " x "
         <<  gpuMaxGridSize_[1] << " x "
         <<  gpuMaxGridSize_[2] << "\n";
    info << "  Max Thread Dims         "
         <<  gpuMaxThreadDim_[0] << " x "
         <<  gpuMaxThreadDim_[1] << " x "
         <<  gpuMaxThreadDim_[2] << "\n";
    info << "  Max Threads/Block       "
         <<  gpuMaxThreadsPerBlock_ << "\n";
    info << "  Warp Size               "
         <<  gpuWarpSize_ << "\n";
    info << "  Num Multiprocessors     "
         <<  gpuNumMultiprocessors_ << "\n";
    info << "  Max Concurrent Kernels  "
         <<  gpuMaxConcurrentKernels_;

    return info.str();
}

}

