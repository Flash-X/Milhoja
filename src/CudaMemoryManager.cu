#include "hip/hip_runtime.h"
#ifndef USE_CUDA_BACKEND
#error "This file need not be compiled if the CUDA backend isn't used"
#endif

#include "CudaMemoryManager.h"

#include <stdexcept>
#include <iostream>

#include "OrchestrationLogger.h"
#include "CudaGpuEnvironment.h"

namespace orchestration {

std::size_t   CudaMemoryManager::nBytes_ = 0;
bool          CudaMemoryManager::instantiated_ = false;

/**
 *
 * \return 
 */
void CudaMemoryManager::instantiate(const std::size_t nBytesInMemoryPools) {
    Logger::instance().log("[CudaMemoryManager] Initializing...");

    if (instantiated_) {
        throw std::logic_error("[CudaMemoryManager::instantiate] "
                               "Memory manager already instantiated");
    } else if (nBytesInMemoryPools == 0) {
        throw std::invalid_argument("[CudaMemoryManager::instantiate] "
                                    "Buffers must be non-empty");
    }
    // TODO: Check that buffers are sized for byte alignment?

    nBytes_ = nBytesInMemoryPools;
    instantiated_ = true;

    instance();

    Logger::instance().log("[CudaMemoryManager] Created and ready for use");
}

/**
 *
 * \return 
 */
CudaMemoryManager&   CudaMemoryManager::instance(void) {
    if (!instantiated_) {
        throw std::logic_error("[CudaMemoryManager::instance] Instantiate first");
    }

    static CudaMemoryManager   manager;
    return manager;
}

/**
 * 
 *
 * \return 
 */
CudaMemoryManager::CudaMemoryManager(void)
    : pinnedBuffer_{nullptr},
      gpuBuffer_{nullptr},
      pinnedOffset_{0},
      gpuOffset_{0}
{
    std::size_t   gpuMemBytes = CudaGpuEnvironment::instance().bytesInDeviceMemory();
    Logger::instance().log(  "[CudaMemoryManager] GPU memory has " 
                           + std::to_string(gpuMemBytes / std::pow(1024.0, 3.0))
                           + " Gb");
    if (nBytes_ >= gpuMemBytes) {
        throw std::invalid_argument("[CudaMemoryManager::CudaMemoryManager] "
                                    "Cannot use all GPU memory as buffer");
    }
    // TODO: How to get RAM size in portable way?

    pthread_cond_init(&memoryReleased_, NULL);
    pthread_mutex_init(&mutex_, NULL);

    pthread_mutex_lock(&mutex_);

    hipError_t    cErr = hipHostMalloc(&pinnedBuffer_, nBytes_);
    if ((cErr != hipSuccess) || (pinnedBuffer_ == nullptr)) {
        pthread_mutex_unlock(&mutex_);
        std::string  errMsg = "[CudaMemoryManager::CudaMemoryManager] ";
        errMsg += "Unable to allocate pinned memory\n";
        errMsg += "Cuda error - " + std::string(hipGetErrorName(cErr)) + "\n";
        errMsg += std::string(hipGetErrorString(cErr)) + "\n";
        throw std::runtime_error(errMsg);
    }
    Logger::instance().log(  "[CudaMemoryManager] Allocated " 
                           + std::to_string(nBytes_ / std::pow(1024.0, 3.0))
                           + " Gb of pinned memory");

    cErr = hipMalloc(&gpuBuffer_, nBytes_);
    if ((cErr != hipSuccess) || (gpuBuffer_ == nullptr)) {
        pthread_mutex_unlock(&mutex_);
        std::string  errMsg = "[CudaMemoryManager::CudaMemoryManager] ";
        errMsg += "Unable to allocate GPU memory\n";
        errMsg += "Cuda error - " + std::string(hipGetErrorName(cErr)) + "\n";
        errMsg += std::string(hipGetErrorString(cErr)) + "\n";
        throw std::runtime_error(errMsg);
    }
    Logger::instance().log(  "[CudaMemoryManager] Allocated " 
                           + std::to_string(nBytes_ / std::pow(1024.0, 3.0))
                           + " Gb of GPU memory");

    pthread_mutex_unlock(&mutex_);
}

/**
 * 
 *
 * \return 
 */
CudaMemoryManager::~CudaMemoryManager(void) {
    Logger::instance().log("[CudaMemoryManager] Finalizing...");

    pthread_mutex_lock(&mutex_);

    if (pinnedBuffer_ != nullptr) {
        hipError_t   cErr = hipHostFree(pinnedBuffer_);
        if (cErr != hipSuccess) {
            std::cerr << "[CudaMemoryManager::~CudaMemoryManager] "
                      << "Unable to deallocate pinned memory\n"
                      << "CUDA error - "
                      << hipGetErrorName(cErr) << "\n"
                      << hipGetErrorString(cErr) << std::endl;
        }
        pinnedBuffer_ = nullptr;
        Logger::instance().log(  "[CudaMemoryManager] Deallocated " 
                               + std::to_string(nBytes_ / std::pow(1024.0, 3.0))
                               + " Gb of pinned memory");
    }

    if (gpuBuffer_ != nullptr) {
        hipError_t   cErr = hipFree(gpuBuffer_);
        if (cErr != hipSuccess) {
            std::cerr << "[CudaMemoryManager::~CudaMemoryManager] "
                      << "Unable to deallocate GPU memory\n"
                      << "CUDA error - "
                      << hipGetErrorName(cErr) << "\n"
                      << hipGetErrorString(cErr) << std::endl;
        }
        gpuBuffer_ = nullptr;
        Logger::instance().log(  "[CudaMemoryManager] Deallocated " 
                               + std::to_string(nBytes_ / std::pow(1024.0, 3.0))
                               + " Gb of GPU memory");
    }
 
    pinnedOffset_ = 0;
    gpuOffset_ = 0;
    instantiated_ = false;

    pthread_mutex_unlock(&mutex_);

    pthread_cond_destroy(&memoryReleased_);
    pthread_mutex_destroy(&mutex_);

    Logger::instance().log("[CudaMemoryManager] Destroyed");
}

/**
 * 
 *
 * \return 
 */
void   CudaMemoryManager::reset(void) {
    // There is no mechanism for now for releasing memory on a per request
    // basis.  This just resets the entire object so that it appears that 
    // no memory has been given out yet.
    pthread_mutex_lock(&mutex_);
    pinnedOffset_ = 0;
    gpuOffset_ = 0;
    pthread_mutex_unlock(&mutex_);

//    Logger::instance().log("[CudaMemoryManager] Reset memory resources");
}

/**
 * Refer to the documentation of the requestGpuMemory function of Backend for
 * more information.
 *
 * @todo Confirm that the request is inline with byte alignment?
 */
void  CudaMemoryManager::requestMemory(const std::size_t pinnedBytes,
                                       void** pinnedPtr,
                                       const std::size_t gpuBytes,
                                       void** gpuPtr) {
    if ((pinnedBytes == 0) || (gpuBytes == 0)) {
        std::string  errMsg = "[CudaMemoryManager::requestMemory] ";
        errMsg += "Requests of zero indicate logical error\n";
        throw std::invalid_argument(errMsg);
    }

    pthread_mutex_lock(&mutex_);

    if ((pinnedOffset_ + pinnedBytes) > nBytes_) {
        pthread_mutex_unlock(&mutex_);
        std::string  errMsg = "[CudaMemoryManager::requestMemory] ";
        errMsg += "Pinned buffer overflow\n";
        errMsg += std::to_string(nBytes_ - pinnedOffset_);
        errMsg += " bytes available and ";
        errMsg += std::to_string(pinnedBytes);
        errMsg += " bytes requested";
        throw std::overflow_error(errMsg);
    } else if ((gpuOffset_ + gpuBytes) > nBytes_) {
        pthread_mutex_unlock(&mutex_);
        std::string  errMsg = "[CudaMemoryManager::requestMemory] ";
        errMsg += "GPU buffer overflow\n";
        errMsg += std::to_string(nBytes_ - gpuOffset_);
        errMsg += " bytes available and ";
        errMsg += std::to_string(gpuBytes);
        errMsg += " bytes requested";
        throw std::overflow_error(errMsg);
    }

    *pinnedPtr = static_cast<void*>(pinnedBuffer_ + pinnedOffset_);
    *gpuPtr    = static_cast<void*>(gpuBuffer_    + gpuOffset_);
    pinnedOffset_ += pinnedBytes;
    gpuOffset_    += gpuBytes;

    pthread_mutex_unlock(&mutex_);
}

/**
 * Refer to the documentation of the releaseGpuMemory function of Backend for
 * more information.
 */
void  CudaMemoryManager::releaseMemory(void** pinnedPtr, void** gpuPtr) {
    // Null so that we don't have dangling pointers.  This is inline with
    // the present reset() ugliness --- at the end of a runtime execution cycle,
    // all data packets should have called this routine so that effectively
    // none of the memory in the pools is checked out.
    *pinnedPtr = nullptr;
    *gpuPtr    = nullptr;
}

}

