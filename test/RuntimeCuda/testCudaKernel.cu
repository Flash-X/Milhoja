#include "hip/hip_runtime.h"
#include "Grid_Axis.h"
#include "Grid_Edge.h"
#include "Grid.h"
#include "RuntimeAction.h"
#include "CudaRuntime.h"
#include "CudaStreamManager.h"
#include "CudaMemoryManager.h"

#include "Flash.h"
#include "constants.h"

#include "setInitialConditions_block.h"
#include "computeLaplacianDensity.h"
#include "computeLaplacianEnergy.h"
#include "scaleEnergy.h"
#include "Analysis.h"

int   main(int argc, char* argv[]) {
    // It appears that OpenACC on Summit with PGI has max 32 asynchronous
    // queues.  If you assign more CUDA streams to queues with OpenACC, then
    // these streams just roll over and the last 32 CUDA streams will be the
    // only streams mapped to queues.
    constexpr int            N_STREAMS = 32; 
    constexpr unsigned int   N_THREAD_TEAMS = 1;
    constexpr unsigned int   MAX_THREADS = 6;
    constexpr std::size_t    MEMORY_POOL_SIZE_BYTES = 4294967296; 
    constexpr std::size_t    N_BLOCKS = N_BLOCKS_X * N_BLOCKS_Y * N_BLOCKS_Z;

    using namespace orchestration;

    CudaRuntime::setNumberThreadTeams(N_THREAD_TEAMS);
    CudaRuntime::setMaxThreadsPerTeam(MAX_THREADS);
    CudaRuntime::setLogFilename("DeleteMe.log");
    std::cout << "\n";
    std::cout << "----------------------------------------------------------\n";
    CudaRuntime::instance().printGpuInformation();
    std::cout << "----------------------------------------------------------\n";
    std::cout << std::endl;

    CudaStreamManager::setMaxNumberStreams(N_STREAMS);
    CudaMemoryManager::setBufferSize(MEMORY_POOL_SIZE_BYTES);

    //***** SET INITIAL CONDITIONS
    // Initialize Grid unit/AMReX
    Grid::instantiate();
    Grid&    grid = Grid::instance();
    grid.initDomain(Simulation::setInitialConditions_block);

    //***** FIRST RUNTIME EXECUTION CYCLE
    RuntimeAction    computeLaplacianDensity_packet;
    computeLaplacianDensity_packet.nInitialThreads = 6;
    computeLaplacianDensity_packet.teamType = ThreadTeamDataType::SET_OF_BLOCKS;
    computeLaplacianDensity_packet.nTilesPerPacket = 1;
    computeLaplacianDensity_packet.routine = ActionRoutines::computeLaplacianDensity_packet_oacc_summit;

    RuntimeAction    computeLaplacianEnergy_packet;
    computeLaplacianEnergy_packet.nInitialThreads = 6;
    computeLaplacianEnergy_packet.teamType = ThreadTeamDataType::SET_OF_BLOCKS;
    computeLaplacianEnergy_packet.nTilesPerPacket = 1;
    computeLaplacianEnergy_packet.routine = ActionRoutines::computeLaplacianEnergy_packet_oacc_summit;

    CudaRuntime::instance().executeGpuTasks("Density", computeLaplacianDensity_packet);
    CudaRuntime::instance().executeGpuTasks("Energy",  computeLaplacianEnergy_packet);

    // TODO: Where to get this value from?  It should be a runtime parameter.
    constexpr Real   ENERGY_SCALE_FACTOR = 5.0;
    std::unique_ptr<Tile>    tileDesc{};
    for (auto ti = grid.buildTileIter(0); ti->isValid(); ti->next()) {
        tileDesc = ti->buildCurrentTile();

        const int        level   = tileDesc->level();
        const IntVect    lo      = tileDesc->lo();
        const IntVect    hi      = tileDesc->hi();
        FArray4D         f       = tileDesc->data();
        const FArray1D   xCoords = grid.getCellCoords(Axis::I, Edge::Center,
                                                      level, lo, hi); 
        const FArray1D   yCoords = grid.getCellCoords(Axis::J, Edge::Center,
                                                      level, lo, hi); 
       
        StaticPhysicsRoutines::scaleEnergy(lo, hi, xCoords, yCoords, f,
                                           ENERGY_SCALE_FACTOR);
    }

    //***** ANALYSIS RUNTIME EXECUTION CYCLE
    RuntimeAction    computeError_block;
    computeError_block.nInitialThreads     = 6;
    computeError_block.teamType            = ThreadTeamDataType::BLOCK;
    computeError_block.nTilesPerPacket     = 0;
    computeError_block.routine             = Analysis::computeErrors_block;

    Analysis::initialize(N_BLOCKS);
    CudaRuntime::instance().executeCpuTasks("Analysis", computeError_block);

    double L_inf1      = 0.0;
    double meanAbsErr1 = 0.0;
    double L_inf2      = 0.0;
    double meanAbsErr2 = 0.0;
    Analysis::densityErrors(&L_inf1, &meanAbsErr1);
    Analysis::energyErrors(&L_inf2, &meanAbsErr2);
    std::cout << "L_inf1 = " << L_inf1 << "\n";
    std::cout << "L_inf2 = " << L_inf2 << std::endl;

//    EXPECT_TRUE(0.0 <= L_inf1);
//    EXPECT_TRUE(L_inf1 <= 1.0e-15);
//    EXPECT_TRUE(0.0 <= meanAbsErr1);
//    EXPECT_TRUE(meanAbsErr1 <= 1.0e-15);

//    EXPECT_TRUE(0.0 <= L_inf2);
//    EXPECT_TRUE(L_inf2 <= 9.0e-6);
//    EXPECT_TRUE(0.0 <= meanAbsErr2);
//    EXPECT_TRUE(meanAbsErr2 <= 9.0e-6);

    // Clean-up
    grid.destroyDomain();
}

